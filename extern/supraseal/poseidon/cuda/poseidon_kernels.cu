#include "hip/hip_runtime.h"
// Copyright Supranational LLC

#include <assert.h>


#ifdef __CUDA_ARCH__

extern __shared__ fr_t scratchpad[];

__device__ __forceinline__
fr_t pow_5(const fr_t& element) {
    fr_t tmp = sqr(element);
    tmp = sqr(tmp);
    return element * tmp;
}

__device__ __forceinline__
void quintic_s_box(fr_t& element, const fr_t& round_constant) {

    element = pow_5(element);
    element += round_constant;
}

__device__ __forceinline__
void partial_quintic_s_box(fr_t& element) {

    element = pow_5(element);
}

__device__ __forceinline__
void add_full_round_constants(fr_t& element, const fr_t& round_constant) {

    element += round_constant;
}

__device__ __forceinline__
void matrix_mul(fr_t& element, const fr_t* matrix, const int t,
                const int thread_pos, const int shared_pos) {

    scratchpad[threadIdx.x] = element;
    __syncthreads();

    element = fr_t::dot_product(&scratchpad[shared_pos], &matrix[thread_pos], t, t);
    __syncthreads();
}

__device__ __forceinline__
fr_t last_matrix_mul(const fr_t* elements, const fr_t* matrix, const int t) {

    return fr_t::dot_product(elements, &matrix[1], t, t);
}

__device__ __forceinline__
void scalar_product(fr_t* elements, const fr_t* sparse_matrix,
                    const int t) {

    elements[0] *= sparse_matrix[0];
    elements[0] += fr_t::dot_product(&elements[1], &sparse_matrix[1], t-1);
}

__device__ __forceinline__
void sparse_matrix_mul(fr_t* elements, const fr_t* sparse_matrix,
                       const int t) {

    fr_t element0 = elements[0];

    scalar_product(elements, sparse_matrix, t);

    #pragma unroll
    for (int i = 1; i < t; i++) {
        elements[i] += element0 * sparse_matrix[t + i - 1];
    }
}

__device__ __forceinline__
void round_matrix_mul(fr_t& element, const kernel_params_t constants,
                      const int current_round, const int thread_pos,
                      const int shared_pos) {

    if (current_round == constants.half_full_rounds - 1) {
        matrix_mul(element, constants.pre_sparse_matrix, constants.t,
                   thread_pos, shared_pos);
    }
    else {
        matrix_mul(element, constants.mds_matrix, constants.t, thread_pos,
                   shared_pos);
    }
}

__device__ __forceinline__
void full_round(fr_t& element, const kernel_params_t constants,
                int& rk_offset, int& current_round, const int thread_pos,
                const int shared_pos) {


    quintic_s_box(element, constants.round_constants[rk_offset]);
    rk_offset += constants.t;

    round_matrix_mul(element, constants, current_round, thread_pos, shared_pos);
    current_round++;
}

__device__ __forceinline__
void partial_round(fr_t* elements, const int t,
                   const kernel_params_t constants,
                   int& rk_offset, int& current_round) {

    quintic_s_box(elements[0], constants.round_constants[rk_offset]);
    rk_offset += 1;

    sparse_matrix_mul(elements, constants.sparse_matrices +
                      (t * 2 - 1) *
                      (current_round - constants.half_full_rounds), t);
    current_round++;
}

__device__ __forceinline__
uint32_t bswap(uint32_t a)
{
  uint32_t ret;
  asm("prmt.b32 %0, %1, %1, 0x0123;" : "=r"(ret) : "r"(a));
  return ret;
}

__device__ __forceinline__
void bswap(fr_t& a)
{
  for (int i = 0; i < a.len(); i++) {
    a[i] = bswap(a[i]);
  }
}

#endif

// Perform first 4 full rounds
//   in_ptrs       - input data
//   aux_ptr       - aux buffer to store results
//   constants     - constants related to configuration & application
//   mont          - if true convert field elements to montgomery form
//   first         - if true this is the first operation on data
//   multi_in_ptr  - if true multiple input pointers are used
// Launch parameters
//   One thread per element (including domain tag)
// in_ptr layout
//   Contains input field elements with one empty element for the
//   domain tag before each 'arity' set of inputs.
//   dt0 fr0 fr1 fr2 fr3 fr4 fr5 fr6 fr7 dt1 etc
//   If multi_in_ptr == false then all field elements are in a contiguous buffer.
//   If multi_in_ptr == true then ARITY pointers are provided, one per branch with data
//   layed out as:
//     in_ptr[0] = s0n0 s1n0 s2n0 s3n0 ...
//     in_ptr[1] = s0n1 s1n1 s2n1 s3n1 ...
//     in_ptr[2] = s0n2 s1n2 s2n2 s3n2 ...
//       ...
//     in_ptr[7] = s0n7 s1n7 s2n7 s3n7 ...
//
// aux_ptr
//   Will contain the hashed outputs in the same layout as in_ptr and should contain
//   space for num_hashes * ARITY_DT elements.
//
// num_hashes - number of inputs to hash
// stride     - number of elements between subsequent inputs to a hash
template<int ARITY_DT> __global__ 
void poseidon_hash_1_0(in_ptrs_d<ARITY_DT - 1> in_ptrs, fr_t* aux_ptr, const fr_t domain_tag,
                       const kernel_params_t constants,
                       const int num_hashes, const int stride,
                       const bool to_mont, const bool do_bswap,
                       const bool first_tree_c, const bool first_tree_r,
                       const bool multi_in_ptr) {
#ifdef __CUDA_ARCH__
  const int ARITY   = ARITY_DT - 1;
  int current_round = 0;
  int rk_offset     = 0;
  
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid >= num_hashes * ARITY_DT) {
    return;
  }

  // Position in shared memory
  int shared_pos = (threadIdx.x / ARITY_DT) * ARITY_DT;
  // Index into set of t elements
  //int idx = blockIdx.x * (blockDim.x / ARITY_DT) + threadIdx.x / ARITY_DT;
  //int idx = tid / ARITY_DT;
  
  // For PC2, traversal is
  // thr0 - dt0
  // thr1 - s0n0l0
  // thr2 - s0n0l1
  // thr3 - s0n0l2
  // ...
  // thr11 - s0n0l10
  // thr12 - s0n1l0
  // thr13 - s0n1l2

  int num_batches = (num_hashes + stride - 1) / stride;
  int hash_num = tid / ARITY_DT;
  // Position within set of t elements
  int hash_input = tid % ARITY_DT;

  int node = hash_num % num_batches;
  int sector = hash_num / num_batches;
  fr_t element;

  if (hash_input == 0) {
    element = domain_tag;
  }
  else {
    if (multi_in_ptr) {
      // This is a bit complicated due to the pattern of the data. When num_hashes
      // is equal to batch size then the pattern is:
      //     in_ptr[0] = s0n0 s1n0 s2n0 s3n0 ...
      //     in_ptr[1] = s0n1 s1n1 s2n1 s3n1 ...
      //     in_ptr[2] = s0n2 s1n2 s2n2 s3n2 ...
      //     in_ptr[7] = s0n7 s1n7 s2n7 s3n7 ...
      // When num_hashes is a multiple of stride, say 2x:
      //     in_ptr[0] = s0n0 s0n1
      //     in_ptr[1] = s0n2 s0n3
      //     in_ptr[2] = s0n4 s0n5
      //     in_ptr[3] = s0n6 s0n7
      //     in_ptr[4] = s1n0 s1n1
      //     in_ptr[7] = s1n6 s1n7
      // ie, two consecutive elements at a time. This pattern is repeated many times
      // based on batch size.

      int elements_per_hash_per_ptr = num_hashes / stride;
      int element_idx = hash_num * (ARITY_DT - 1) + (hash_input - 1);
      int element_batch_idx = element_idx / elements_per_hash_per_ptr;
      int element_batch_off = element_idx % elements_per_hash_per_ptr;
      int ptr_num = element_batch_idx % ARITY;
      int ptr_idx = element_idx / (ARITY * elements_per_hash_per_ptr) * elements_per_hash_per_ptr + element_batch_off;
      element = in_ptrs.ptrs[ptr_num][ptr_idx];
    } else {
      fr_t* in_ptr = in_ptrs.ptrs[0];
      if (first_tree_r || first_tree_c) {
        int first_element;
        int element_index;
        if (first_tree_c) {
          first_element = node * stride + sector;
          element_index = first_element + (hash_input - 1) * stride * num_batches;
        } else {
          first_element = node * ARITY * stride + sector;
          element_index = first_element + (hash_input - 1) * stride;
        }
        element = in_ptr[element_index];
        if (do_bswap) {
          bswap(element);
        }
      } else {
        // Access element from a packed array (no domain tag)
        element = in_ptr[hash_num * (ARITY_DT - 1) + (hash_input - 1)];
      }
    }
    if (to_mont) {
      element.to();
    }
  }

  rk_offset += hash_input;

  add_full_round_constants(element, constants.round_constants[rk_offset]);
  rk_offset += ARITY_DT;

  for (int i = 0; i < constants.half_full_rounds; i++) {
    full_round(element, constants, rk_offset, current_round, hash_input,
               shared_pos);
  }

  // When first is true this unstrides the sectors from pc1, leading to
  // s0n0 s0n1 ... s1n0 s1n1 ... s2n0 s2n1 ...
  
  __syncthreads();
  aux_ptr[hash_num * ARITY_DT + hash_input] = element;
#endif
}

// Perform partial rounds
// Data is in aux_ptr from poseidon_hash_1
// rk_offset     - 5 * t
// current_round - 4
// Launch params
//   One thread per t elements
template<int ARITY_DT> __global__ 
void poseidon_hash_2(fr_t* aux_ptr, const kernel_params_t constants,
                     int rk_offset, int current_round, const int batch_size) {

#ifdef __CUDA_ARCH__
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= batch_size) {
    return;
  }
  
  aux_ptr += idx * ARITY_DT;

  fr_t elements[ARITY_DT];

  for (int i = 0; i < ARITY_DT; i++) {
    elements[i] = aux_ptr[i];
  }

  for (int i = 0; i < constants.partial_rounds; i++) {
    partial_round(elements, ARITY_DT, constants, rk_offset, current_round);
  }

  for (int i = 0; i < ARITY_DT; i++) {
    aux_ptr[i] = elements[i];
  }
#endif
}

// Perform 3 of the final 4 full rounds
// rk_offset - 5 * t + number of partial rounds for this config from partial_rounds_map
// current_round - 4 + number of partial rounds for this config from partial_rounds_map
// Launch parameters
//   One thread per element (including domain tag)
template<int ARITY_DT> __global__ 
void poseidon_hash_3(fr_t* aux_ptr, const kernel_params_t constants,
                     int rk_offset, int current_round, const int batch_size) {
#ifdef __CUDA_ARCH__
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= batch_size) {
    return;
  }

  int thread_pos = threadIdx.x % ARITY_DT;
  int shared_pos = (threadIdx.x / ARITY_DT) * ARITY_DT;
  idx = blockIdx.x * (blockDim.x / ARITY_DT) + threadIdx.x / ARITY_DT;

  rk_offset += thread_pos;

  fr_t element = aux_ptr[idx * ARITY_DT + thread_pos];

  for (int i = 0; i < constants.half_full_rounds - 1; i++) {
    full_round(element, constants, rk_offset, current_round, thread_pos,
               shared_pos);
  }

  partial_quintic_s_box(element);

  aux_ptr[idx * ARITY_DT + thread_pos] = element;
#endif
}

// Perform last of the final 4 full rounds
// Data is in aux_ptr from poseidon_hash_1
// Output is written to out_ptr
// Launch params
//   One thread per t elements
template<int ARITY_DT> __global__ 
void poseidon_hash_4(const fr_t* aux_ptr, fr_t* out_ptr, const fr_t* mds_matrix,
                     const int batch_size, const bool from_mont) {
#ifdef __CUDA_ARCH__
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= batch_size) {
    return;
  }

  aux_ptr += idx * ARITY_DT;
  // fr_t elements[t];

  // for (int i = 0; i < t; i++) {
  //   elements[i] = aux_ptr[i];
  // }

  // This writes state[1] into out_ptr
  // No dt slots
  fr_t out = last_matrix_mul(aux_ptr, mds_matrix, ARITY_DT);
  if (from_mont) {
    out.from();
  }
  out_ptr[idx] = out;
#endif
}
