#include "hip/hip_runtime.h"
// Copyright Supranational LLC

#include <ntt/ntt.cuh>

__launch_bounds__(1024)
__global__ void coeff_wise_mult(fr_t* a, const fr_t* b, uint32_t lg_domain_size)
{
    uint32_t idx0 = threadIdx.x + blockIdx.x * blockDim.x;
    size_t limit = (size_t)1 << lg_domain_size;

    for (size_t idx = idx0; idx < limit; idx += blockDim.x * gridDim.x)
        a[idx] *= b[idx];
}

__launch_bounds__(1024)
__global__ void sub_mult_with_constant(fr_t* a, const fr_t* c, fr_t z,
                                       uint32_t lg_domain_size)
{
    uint32_t idx0 = threadIdx.x + blockIdx.x * blockDim.x;
    size_t limit = (size_t)1 << lg_domain_size;

    for (size_t idx = idx0; idx < limit; idx += blockDim.x * gridDim.x) {
        fr_t r = a[idx] - c[idx];
        a[idx] = r * z;
    }
}

#ifndef __CUDA_ARCH__

const size_t gib = (size_t)1 << 30;

class ntt_msm_h : public NTT {
private:
    static fr_t calculate_z_inv(size_t lg_domain_size) {
        fr_t gen_pow = group_gen;
        while (lg_domain_size--)
            gen_pow ^= 2;
        return (gen_pow - fr_t::one()).reciprocal();
    }

    static void execute_ntts_single(fr_t* d_inout, const fr_t* in,
                                    size_t lg_domain_size, size_t actual_size,
                                    stream_t& stream)
    {
        size_t domain_size = (size_t)1 << lg_domain_size;

        assert(actual_size <= domain_size);

        stream.HtoD(&d_inout[0], in, actual_size);

        if (actual_size < domain_size) {
            hipMemsetAsync(&d_inout[actual_size], 0,
                (domain_size - actual_size) * sizeof(fr_t), stream);
        }

        NTT_internal(&d_inout[0], lg_domain_size,
            NTT::InputOutputOrder::NR, NTT::Direction::inverse,
            NTT::Type::standard, stream);
        NTT_internal(&d_inout[0], lg_domain_size,
            NTT::InputOutputOrder::RN, NTT::Direction::forward,
            NTT::Type::coset, stream);
    }

    static int lg2(size_t n)
    {   int ret = 0; while (n >>= 1) ret++; return ret;   }

public:

    // a, b, c = coset_ntt(intt(a, b, c))
    // a *= b
    // a -= c
    // a[i] /= (multiplicative_gen^domain_size) - 1
    // a = coset_intt(a)
    // a is the result vector
    static void execute_ntt_msm_h(const gpu_t& gpu, gpu_ptr_t<fr_t> d_a,
                                  const Assignment<fr_t>& input,
                                  slice_t<affine_t> points_h,
                                  point_t& result_h)
    {
        size_t actual_size = input.abc_size;
        size_t npoints = points_h.size();
        size_t lg_domain_size = lg2(npoints - 1) + 1;
        size_t domain_size = (size_t)1 << lg_domain_size;

        fr_t z_inv = calculate_z_inv(lg_domain_size);

        int sm_count = gpu.props().multiProcessorCount;

        bool lot_of_memory = 3 * domain_size * sizeof(fr_t) <
                             gpu.props().totalGlobalMem - gib;
        {
            dev_ptr_t<fr_t> d_b(domain_size * (lot_of_memory + 1));
            fr_t* d_c = &d_b[domain_size * lot_of_memory];

            event_t sync_event;

            execute_ntts_single(&d_a[0], input.a, lg_domain_size,
                                actual_size, gpu[0]);
            sync_event.record(gpu[0]);

            execute_ntts_single(&d_b[0], input.b, lg_domain_size,
                                actual_size, gpu[1]);

            sync_event.wait(gpu[1]);
            coeff_wise_mult<<<sm_count, 1024, 0, gpu[1]>>>
                (&d_a[0], &d_b[0], (index_t)lg_domain_size);
            sync_event.record(gpu[1]);

            execute_ntts_single(&d_c[0], input.c, lg_domain_size,
                                actual_size, gpu[1 + lot_of_memory]);

            sync_event.wait(gpu[1 + lot_of_memory]);
            sub_mult_with_constant<<<sm_count, 1024, 0, gpu[1 + lot_of_memory]>>>
                (&d_a[0], &d_c[0], z_inv, (index_t)lg_domain_size);
        }

        NTT_internal(&d_a[0], lg_domain_size, NTT::InputOutputOrder::NN,
            NTT::Direction::inverse, NTT::Type::coset, gpu[1 + lot_of_memory]);

        gpu[1 + lot_of_memory].sync();

        msm_t<bucket_t, point_t, affine_t, fr_t> msm(nullptr, npoints);
        msm.invoke(result_h, points_h, d_a, true);
    }
};

#endif
