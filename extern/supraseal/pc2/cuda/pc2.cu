#include "hip/hip_runtime.h"
// Copyright Supranational LLC

#include "../../poseidon/cuda/poseidon.cu"
#include "../../util/debug_helpers.hpp"
#include "host_ptr_t.hpp"

#ifndef __CUDA_ARCH__

#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/mman.h>

#include <filesystem>
#include <chrono>
#include "../planner.cpp"
#include "pc2.cuh"
#include "cuda_lambda_t.hpp"
#include "../../util/util.hpp"

template<class C>
pc2_t<C>::pc2_t(topology_t& _topology,
                   bool _tree_r_only, streaming_node_reader_t<C>& _reader,
                   size_t _nodes_to_read, size_t _batch_size,
                   size_t _stream_count,
                   const char** _data_filenames, const char* _output_dir) :
  topology(_topology),
  tree_r_only(_tree_r_only),
  reader(_reader),
  nodes_to_read(_nodes_to_read),
  batch_size(_batch_size),
  tree_c_address(C::GetNumNodes() / C::GetNumTreeRCFiles(),
                 C::GetNumTreeRCArity(), NODE_SIZE, 0),
  tree_r_address(C::GetNumNodes() / C::GetNumTreeRCFiles(),
                 C::GetNumTreeRCArity(), NODE_SIZE, C::GetNumTreeRDiscardRows() + 1),
  stream_count(_stream_count),
  tree_c_partition_roots(C::PARALLEL_SECTORS * C::GetNumTreeRCFiles()),
  tree_r_partition_roots(C::PARALLEL_SECTORS * C::GetNumTreeRCFiles()),
  gpu_results_c(tree_r_only ? 0 :_batch_size * C::PARALLEL_SECTORS / C::GetNumTreeRCArity() * stream_count),
  gpu_results_r(_batch_size * C::PARALLEL_SECTORS / C::GetNumTreeRCArity() * stream_count),
  host_buf_storage(num_host_bufs * batch_size * C::PARALLEL_SECTORS),
  data_filenames(_data_filenames),
  output_dir(_output_dir)
{
  assert (C::GetNumTreeRCArity() == C::GetNumTreeRCArity());
  assert (nodes_to_read % stream_count == 0);

  open_files();

  // Compute the final offset in the file for GPU data
  const size_t cpu_nodes_to_hash = batch_size * stream_count / C::GetNumTreeRCArity() / C::GetNumTreeRCArity();
  tree_address_t<C> final_tree(cpu_nodes_to_hash, C::GetNumTreeRCArity(), sizeof(fr_t), 0);
  final_gpu_offset_c = tree_c_address.data_size() - final_tree.data_size();
  final_gpu_offset_r = tree_r_address.data_size() - final_tree.data_size();

  // Compute an offset table used for multiple partitions
  size_t nodes_per_stream = nodes_to_read / stream_count;
  size_t layer_offset = nodes_per_stream;
  while (layer_offset >= C::GetNumTreeRCArity()) {
    layer_offsets_c.push_back(layer_offset);
    layer_offset /= C::GetNumTreeRCArity();
  }

  layer_offset = nodes_per_stream;
  for (size_t i = 0; i < C::GetNumTreeRDiscardRows() + 1; i++) {
    layer_offset /= C::GetNumTreeRCArity();
  }
  while (layer_offset >= C::GetNumTreeRCArity()) {
    layer_offsets_r.push_back(layer_offset);
    layer_offset /= C::GetNumTreeRCArity();
  }

  if (!tree_r_only)
    poseidon_columns.resize(ngpus());

  // Create GPU poseidon hashers and streams
  size_t resource_id = 0;
  for (size_t i = 0; i < ngpus(); i++) {
    auto& gpu = select_gpu(i);
    if (!tree_r_only) {
      switch (C::GetNumLayers()) {
      case 2:
        poseidon_columns[i].arity_2 = new PoseidonCuda<3>(gpu);
        break;
      case 11:
        poseidon_columns[i].arity_11 = new PoseidonCuda<12>(gpu);
        break;
      default:
        assert(false);
      }
    }
    poseidon_trees.push_back(new PoseidonCuda<C::GetNumTreeRCArityDT()>(gpu));

    for (size_t j = 0; j < stream_count / ngpus(); j++) {
      resources.push_back(new gpu_resource_t<C>(resource_id, gpu,
                                                   nodes_per_stream, batch_size));
      resource_id++;
    }
  }

  // Register the SPDK page buffer with the CUDA driver
  size_t page_buffer_size = 0;
  page_buffer = (uint8_t*)reader.get_full_buffer(page_buffer_size);
  hipHostRegister(page_buffer, page_buffer_size, hipHostRegisterDefault);

  // Set up host side buffers for returning data
  host_bufs.resize(num_host_batches * disk_io_batch_size);
  host_batches.resize(num_host_batches + num_host_empty_batches);
  host_buf_pool_full.create(num_host_batches + num_host_empty_batches);
  host_buf_pool_empty.create(num_host_batches + num_host_empty_batches);
  host_buf_to_disk.create(num_host_batches + num_host_empty_batches);

  for (size_t i = 0; i < num_host_batches; i++) {
    for (size_t j = 0; j < disk_io_batch_size; j++) {
      host_batches[i].batch[j] = &host_bufs[i * disk_io_batch_size + j];
      host_batches[i].batch[j]->data =
        &host_buf_storage[i * disk_io_batch_size * batch_size * C::PARALLEL_SECTORS +
                          j * batch_size * C::PARALLEL_SECTORS];
    }
    host_buf_pool_full.enqueue(&host_batches[i]);
  }
  for (size_t i = 0; i < num_host_empty_batches; i++) {
    for (size_t j = 0; j < disk_io_batch_size; j++) {
      host_batches[i + num_host_batches].batch[j] = nullptr;
    }
    host_buf_pool_empty.enqueue(&host_batches[i + num_host_batches]);
  }
}

template<class C>
pc2_t<C>::~pc2_t() {
  while (resources.size() > 0) {
    gpu_resource_t<C>* r = resources.back();
    select_gpu(r->gpu);

    delete r;
    resources.pop_back();
  }
  for (size_t i = 0; i < ngpus(); i++) {
    if (!tree_r_only) {
      switch (C::GetNumLayers()) {
        case 2:
          delete poseidon_columns[i].arity_2;
          break;
        case 11:
          delete poseidon_columns[i].arity_11;
          break;
      }
    }
    delete poseidon_trees[i];
  }
  for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
    for (auto it : tree_c_files[i]) {
      delete it;
    }
    for (auto it : tree_r_files[i]) {
      delete it;
    }
  }
  hipHostUnregister(page_buffer);
}
/*
template<class C>
void pc2_t<C>::get_filenames(const char* output_dir,
                                std::vector<std::string>& directories,
                                std::vector<std::string>& p_aux_filenames,
                                std::vector<std::vector<std::string>>& tree_c_filenames,
                                std::vector<std::vector<std::string>>& tree_r_filenames,
                                std::vector<std::string>& sealed_filenames) {
  // Put layer11 / sealed file in a replicas directory if it exists
  std::string pc2_replica_output_dir = output_dir;
  pc2_replica_output_dir += "/replicas";
  if (!std::filesystem::exists(pc2_replica_output_dir.c_str())) {
    pc2_replica_output_dir = output_dir;
  }

  const char* p_aux_template;
  if (C::PARALLEL_SECTORS == 1) {
    p_aux_template = "%s/p_aux";
  } else {
    p_aux_template = "%s/%03ld/p_aux";
  }
  // Open all tree-c and tree-r files
  const char* tree_c_filename_template;
  const char* tree_r_filename_template;
  if (C::PARALLEL_SECTORS == 1) {
    if (C::GetNumTreeRCFiles() > 1) {
      tree_c_filename_template = "%s/sc-02-data-tree-c-%ld.dat";
      tree_r_filename_template = "%s/sc-02-data-tree-r-last-%ld.dat";
    } else {
      tree_c_filename_template = "%s/sc-02-data-tree-c.dat";
      tree_r_filename_template = "%s/sc-02-data-tree-r-last.dat";
    }
  } else {
    if (C::GetNumTreeRCFiles() > 1) {
      tree_c_filename_template = "%s/%03ld/sc-02-data-tree-c-%ld.dat";
      tree_r_filename_template = "%s/%03ld/sc-02-data-tree-r-last-%ld.dat";
    } else {
      tree_c_filename_template = "%s/%03ld/sc-02-data-tree-c.dat";
      tree_r_filename_template = "%s/%03ld/sc-02-data-tree-r-last.dat";
    }
  }
  // And sealed files
  const char* sealed_filename_template;
  if (C::PARALLEL_SECTORS == 1) {
    sealed_filename_template = "%s/sealed-file";
  } else {
    sealed_filename_template = "%s/%03ld/sealed-file";
  }

  directories.push_back(output_dir);

  tree_c_filenames.resize(C::PARALLEL_SECTORS);
  tree_r_filenames.resize(C::PARALLEL_SECTORS);

  const size_t MAX = 256;
  char fname[MAX];
  for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
    // Create sector subdirs
    if (C::PARALLEL_SECTORS == 1) {
      snprintf(fname, MAX, "%s", output_dir);
    } else {
      snprintf(fname, MAX, "%s/%03ld", output_dir, i);
    }
    directories.push_back(fname);

    if (C::PARALLEL_SECTORS == 1) {
      snprintf(fname, MAX, p_aux_template, output_dir);
    } else {
      snprintf(fname, MAX, p_aux_template, output_dir, i);
    }
    p_aux_filenames.push_back(fname);

    if (C::PARALLEL_SECTORS == 1) {
      snprintf(fname, MAX, "%s", pc2_replica_output_dir.c_str());
    } else {
      snprintf(fname, MAX, "%s/%03ld", pc2_replica_output_dir.c_str(), i);
    }
    directories.push_back(fname);

    for (size_t j = 0; j < C::GetNumTreeRCFiles(); j++) {
      // tree-c
      if (C::PARALLEL_SECTORS == 1) {
        if (C::GetNumTreeRCFiles() > 1) {
          snprintf(fname, MAX, tree_c_filename_template, output_dir, j);
        } else {
          snprintf(fname, MAX, tree_c_filename_template, output_dir);
        }
      } else {
        if (C::GetNumTreeRCFiles() > 1) {
          snprintf(fname, MAX, tree_c_filename_template, output_dir, i, j);
        } else {
          snprintf(fname, MAX, tree_c_filename_template, output_dir, i);
        }
      }
      tree_c_filenames[i].push_back(fname);

      // tree-r
      if (C::PARALLEL_SECTORS == 1) {
        if (C::GetNumTreeRCFiles() > 1) {
          snprintf(fname, MAX, tree_r_filename_template, output_dir, j);
        } else {
          snprintf(fname, MAX, tree_r_filename_template, output_dir);
        }
      } else {
        if (C::GetNumTreeRCFiles() > 1) {
          snprintf(fname, MAX, tree_r_filename_template, output_dir, i, j);
        } else {
          snprintf(fname, MAX, tree_r_filename_template, output_dir, i);
        }
      }
      tree_r_filenames[i].push_back(fname);
    }

    // Data files for encoding
    if (C::PARALLEL_SECTORS == 1) {
      snprintf(fname, MAX, sealed_filename_template, pc2_replica_output_dir.c_str());
   } else {
      snprintf(fname, MAX, sealed_filename_template, pc2_replica_output_dir.c_str(), i);
    }
    sealed_filenames.push_back(fname);
  }
}
 */

template<class C>
void pc2_t<C>::get_filenames(const char* output_dir,
                             std::vector<std::string>& directories,
                             std::vector<std::string>& p_aux_filenames,
                             std::vector<std::vector<std::string>>& tree_c_filenames,
                             std::vector<std::vector<std::string>>& tree_r_filenames,
                             std::vector<std::string>& sealed_filenames) {
    std::string pc2_replica_output_dir = output_dir;
    pc2_replica_output_dir += "/replicas";
    if (!std::filesystem::exists(pc2_replica_output_dir.c_str())) {
        pc2_replica_output_dir = output_dir;
    }

    if (strncmp(output_dir, "//multi//", 9) == 0) {
        const char* custom_paths = output_dir + 9;
        parse_custom_paths(custom_paths, directories, p_aux_filenames, tree_c_filenames, tree_r_filenames, sealed_filenames);
    } else {
        generate_default_paths(output_dir, pc2_replica_output_dir, directories, p_aux_filenames, tree_c_filenames, tree_r_filenames, sealed_filenames);
    }
}

template<class C>
void pc2_t<C>::parse_custom_paths(const char* custom_paths,
                                  std::vector<std::string>& directories,
                                  std::vector<std::string>& p_aux_filenames,
                                  std::vector<std::vector<std::string>>& tree_c_filenames,
                                  std::vector<std::vector<std::string>>& tree_r_filenames,
                                  std::vector<std::string>& sealed_filenames) {
    const size_t MAX = 256;
    char fname[MAX];

    tree_c_filenames.resize(C::PARALLEL_SECTORS);
    tree_r_filenames.resize(C::PARALLEL_SECTORS);

    for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
        uint32_t len;
        memcpy(&len, custom_paths, sizeof(len));
        custom_paths += sizeof(len);

        std::string replicaPath(custom_paths, len);
        custom_paths += len;
        snprintf(fname, MAX, "%s", replicaPath.c_str());
        sealed_filenames.push_back(fname);

        memcpy(&len, custom_paths, sizeof(len));
        custom_paths += sizeof(len);
        std::string cacheDir(custom_paths, len);
        custom_paths += len;

        directories.push_back(cacheDir);

        snprintf(fname, MAX, "%s/p_aux", cacheDir.c_str());
        p_aux_filenames.push_back(fname);

        for (size_t j = 0; j < C::GetNumTreeRCFiles(); j++) {
            snprintf(fname, MAX, "%s/sc-02-data-tree-c-%ld.dat", cacheDir.c_str(), j);
            tree_c_filenames[i].push_back(fname); ////

            snprintf(fname, MAX, "%s/sc-02-data-tree-r-last-%ld.dat", cacheDir.c_str(), j);
            tree_r_filenames[i].push_back(fname);
        }
    }
}

template<class C>
void pc2_t<C>::generate_default_paths(const char* output_dir,
                                      const std::string& pc2_replica_output_dir,
                                      std::vector<std::string>& directories,
                                      std::vector<std::string>& p_aux_filenames,
                                      std::vector<std::vector<std::string>>& tree_c_filenames,
                                      std::vector<std::vector<std::string>>& tree_r_filenames,
                                      std::vector<std::string>& sealed_filenames) {
    directories.push_back(output_dir);

    tree_c_filenames.resize(C::PARALLEL_SECTORS);
    tree_r_filenames.resize(C::PARALLEL_SECTORS);

    for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
        add_paths_for_sector(output_dir, i, pc2_replica_output_dir, directories, p_aux_filenames, tree_c_filenames, tree_r_filenames, sealed_filenames);
    }
}

template<class C>
void pc2_t<C>::add_paths_for_sector(const char* output_dir,
                                    size_t sector,
                                    const std::string& pc2_replica_output_dir,
                                    std::vector<std::string>& directories,
                                    std::vector<std::string>& p_aux_filenames,
                                    std::vector<std::vector<std::string>>& tree_c_filenames,
                                    std::vector<std::vector<std::string>>& tree_r_filenames,
                                    std::vector<std::string>& sealed_filenames) {
    const size_t MAX = 256;
    char fname[MAX];

    const char* p_aux_template = (C::PARALLEL_SECTORS == 1) ? "%s/p_aux" : "%s/%03ld/p_aux";
    const char* tree_c_filename_template = (C::PARALLEL_SECTORS == 1) ?
                                            (C::GetNumTreeRCFiles() > 1 ? "%s/sc-02-data-tree-c-%ld.dat" : "%s/sc-02-data-tree-c.dat") :
                                            (C::GetNumTreeRCFiles() > 1 ? "%s/%03ld/sc-02-data-tree-c-%ld.dat" : "%s/%03ld/sc-02-data-tree-c.dat");
    const char* tree_r_filename_template = (C::PARALLEL_SECTORS == 1) ?
                                            (C::GetNumTreeRCFiles() > 1 ? "%s/sc-02-data-tree-r-last-%ld.dat" : "%s/sc-02-data-tree-r-last.dat") :
                                            (C::GetNumTreeRCFiles() > 1 ? "%s/%03ld/sc-02-data-tree-r-last-%ld.dat" : "%s/%03ld/sc-02-data-tree-r-last.dat");
    const char* sealed_filename_template = (C::PARALLEL_SECTORS == 1) ? "%s/sealed-file" : "%s/%03ld/sealed-file";

    if (C::PARALLEL_SECTORS == 1) {
        snprintf(fname, MAX, "%s", output_dir);
    } else {
        snprintf(fname, MAX, "%s/%03ld", output_dir, sector);
    }
    directories.push_back(fname);

    if (C::PARALLEL_SECTORS == 1) {
        snprintf(fname, MAX, p_aux_template, output_dir);
    } else {
        snprintf(fname, MAX, p_aux_template, output_dir, sector);
    }
    p_aux_filenames.push_back(fname);

    if (C::PARALLEL_SECTORS == 1) {
        snprintf(fname, MAX, "%s", pc2_replica_output_dir.c_str());
    } else {
        snprintf(fname, MAX, "%s/%03ld", pc2_replica_output_dir.c_str(), sector);
    }
    directories.push_back(fname);

    for (size_t j = 0; j < C::GetNumTreeRCFiles(); j++) {
        if (C::PARALLEL_SECTORS == 1) {
            snprintf(fname, MAX, tree_c_filename_template, output_dir, j);
        } else {
            snprintf(fname, MAX, tree_c_filename_template, output_dir, sector, j);
        }
        tree_c_filenames[sector].push_back(fname);

        if (C::PARALLEL_SECTORS == 1) {
            snprintf(fname, MAX, tree_r_filename_template, output_dir, j);
        } else {
            snprintf(fname, MAX, tree_r_filename_template, output_dir, sector, j);
        }
        tree_r_filenames[sector].push_back(fname);
    }

    if (C::PARALLEL_SECTORS == 1) {
        snprintf(fname, MAX, sealed_filename_template, pc2_replica_output_dir.c_str());
    } else {
        snprintf(fname, MAX, sealed_filename_template, pc2_replica_output_dir.c_str(), sector);
    }
    sealed_filenames.push_back(fname);
}

template<class C>
void pc2_t<C>::open_files() {
  std::vector<std::string> directories;
  std::vector<std::vector<std::string>> tree_c_filenames;
  std::vector<std::vector<std::string>> tree_r_filenames;
  std::vector<std::string> sealed_filenames;

  get_filenames(output_dir,
                directories,
                p_aux_filenames,
                tree_c_filenames,
                tree_r_filenames,
                sealed_filenames);

  for (auto it : directories) {
    if (!std::filesystem::exists(it)) {
      std::filesystem::create_directory(it);
    }
  }
  has_cc_sectors = false;
  has_non_cc_sectors = false;

  size_t num_tree_files = C::GetNumTreeRCFiles();
  for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
    if (!tree_r_only) {
      tree_c_files[i].resize(num_tree_files);
    }
    tree_r_files[i].resize(num_tree_files);
    for (size_t j = 0; j < num_tree_files; j++) {
      // tree-c
      if (!tree_r_only) {
        tree_c_files[i][j] = new file_writer_t<fr_t>();
        assert(tree_c_files[i][j]->open(tree_c_filenames[i][j],
                                        tree_c_address.data_size(), true, false) == 0);
        tree_c_files[i][j]->advise_random();
      }

      // tree-r
      tree_r_files[i][j] = new file_writer_t<fr_t>();
      assert(tree_r_files[i][j]->open(tree_r_filenames[i][j],
                                      tree_r_address.data_size(), true, false) == 0);
      tree_r_files[i][j]->advise_random();
    }

    // Data files for encoding
    if (data_filenames != nullptr && data_filenames[i] != nullptr) {
      data_files[i].mmap_read(data_filenames[i], C::GetSectorSize());
      // If there is a data file present we will encode layer 11 and write the
      // sealed data
      assert(sealed_files[i].open(sealed_filenames[i], C::GetSectorSize(), true, false) == 0);
      has_non_cc_sectors = true;
    } else {
      // Write the raw layer 11 data
      // It would be nice to write different files for encoded vs not encoded data but in
      // reality we can't differentiate between CC and sectors that will use remote data.
      // So we write them all to 'sealed_data' here.
      assert(sealed_files[i].open(sealed_filenames[i], C::GetSectorSize(), true, false) == 0);
      has_cc_sectors = true;
    }
  }
}

template<class C>
void pc2_t<C>::hash() {
  thread_pool_t pool(1);
  pool.spawn([&]() {
    // Affinitize the thread in the pool
    set_core_affinity(topology.pc2_hasher_cpu);
  });

  // Use a channel to prevent the GPU from racing ahead of the CPU
  channel_t<int> ch;
  ch.send(-1);

  host_buffer_t cpu_input_c(gpu_results_c.size());
  host_buffer_t cpu_input_r(gpu_results_r.size());

  auto start = std::chrono::high_resolution_clock::now();
  for (size_t partition = 0; partition < C::GetNumTreeRCFiles(); partition++) {
    auto pstart_gpu = std::chrono::high_resolution_clock::now();
    hash_gpu(partition);
    auto pstop_gpu = std::chrono::high_resolution_clock::now();

    gpu_results_in_use.lock();
    ch.recv();
    pool.spawn([&, partition]() {
      // Protect against a race condition for gpu_results where if the CPU hashing
      // is slow relative to the GPU the results could be overwritten before they are
      // used.
      memcpy(&cpu_input_c[0], &gpu_results_c[0], gpu_results_c.size() * sizeof(fr_t));
      memcpy(&cpu_input_r[0], &gpu_results_r[0], gpu_results_r.size() * sizeof(fr_t));

      gpu_results_in_use.unlock();

      if (!tree_r_only) {
        hash_cpu(&tree_c_partition_roots[partition * C::PARALLEL_SECTORS],
                 partition, &(cpu_input_c[0]), tree_c_files, final_gpu_offset_c);
      }
      hash_cpu(&tree_r_partition_roots[partition * C::PARALLEL_SECTORS],
               partition, &(cpu_input_r[0]), tree_r_files, final_gpu_offset_r);
      ch.send(partition);
    });
    auto pstop_cpu = std::chrono::high_resolution_clock::now();
    uint64_t secs_gpu = std::chrono::duration_cast<
      std::chrono::seconds>(pstop_gpu - pstart_gpu).count();
    uint64_t secs_cpu = std::chrono::duration_cast<
      std::chrono::seconds>(pstop_cpu - pstop_gpu).count();
    printf("Partition %ld took %ld seconds (gpu %ld, cpu %ld)\n",
           partition, secs_gpu + secs_cpu, secs_gpu, secs_cpu);
  }
  ch.recv();
  write_roots(&tree_c_partition_roots[0], &tree_r_partition_roots[0]);
  auto stop = std::chrono::high_resolution_clock::now();
  uint64_t secs = std::chrono::duration_cast<
    std::chrono::seconds>(stop - start).count();

  size_t total_page_reads = nodes_to_read * C::GetNumTreeRCFiles() /
    C::NODES_PER_PAGE * C::GetNumLayers();
  printf("pc2 took %ld seconds utilizing %0.1lf iOPS\n",
         secs, (double)total_page_reads / (double)secs);
}

template<class C>
void pc2_t<C>::process_writes(int core, size_t max_write_size,
                                 mtx_fifo_t<buf_to_disk_batch_t>& to_disk_fifo,
                                 mtx_fifo_t<buf_to_disk_batch_t>& pool,
                                 std::atomic<bool>& terminate,
                                 std::atomic<int>& disk_writer_done) {
  set_core_affinity(core);
  fr_t* staging = new fr_t[max_write_size];

  size_t count = 0;
  while(!terminate || to_disk_fifo.size() > 0) {
    if (pool.is_full()) {
      continue;
    }

    buf_to_disk_batch_t* to_disk_batch = to_disk_fifo.dequeue();
    if (to_disk_batch != nullptr) {
#ifndef DISABLE_FILE_WRITES
      for (size_t batch_elmt = 0; batch_elmt < disk_io_batch_size; batch_elmt++) {
        buf_to_disk_t<C>* to_disk = to_disk_batch->batch[batch_elmt];
        if (to_disk == nullptr || to_disk->size == 0) {
          continue;
        }
        // printf("Writing batch element %ld stride %ld size %ld %p\n",
        //        batch_elmt, to_disk->stride, to_disk->size, to_disk->data);
        if (to_disk->stride == 1) {
          // Copy chunks of contiguous data
          for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
            if (to_disk->src[i] != nullptr) {
              // printf("Writing from %p to %p offset %ld size %ld\n",
              //        to_disk->src[i], to_disk->dst[i], to_disk->offset, to_disk->size);
              to_disk->dst[i]->write_data(to_disk->offset, to_disk->src[i], to_disk->size);
            }
          }
        } else {
          //  Copy strided src data
          assert (max_write_size <= to_disk->size);
          for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
            if (to_disk->src[i] != nullptr) {
              for (size_t j = 0; j < to_disk->size; j++) {
                staging[j] = to_disk->src[i][j * to_disk->stride];
                if (to_disk->reverse) {
                  node_t *n = (node_t*)&staging[j];
                  n->reverse_l();
                }
              }
              to_disk->dst[i]->write_data(to_disk->offset, staging, to_disk->size);
            }
          }
        }
      }
#endif
      //      count++;
      pool.enqueue(to_disk_batch);
    }
  }
  delete [] staging;
  disk_writer_done--;
}

template<class C>
struct pc2_batcher_t {
  typedef typename pc2_t<C>::buf_to_disk_batch_t buf_to_disk_batch_t;

  buf_to_disk_batch_t* unbundle;
  buf_to_disk_batch_t* bundle;
  mtx_fifo_t<buf_to_disk_batch_t>& to_disk;
  mtx_fifo_t<buf_to_disk_batch_t>& pool_full;
  mtx_fifo_t<buf_to_disk_batch_t>& pool_empty;
  size_t idx_unbundle;
  size_t idx_bundle;
  std::mutex mtx;

  pc2_batcher_t(mtx_fifo_t<buf_to_disk_batch_t>& _pool_full,
                mtx_fifo_t<buf_to_disk_batch_t>& _pool_empty,
                mtx_fifo_t<buf_to_disk_batch_t>& _to_disk)
    : pool_full(_pool_full), pool_empty(_pool_empty), to_disk(_to_disk)
  {
    unbundle = pool_full.dequeue();
    bundle = pool_empty.dequeue();
    assert (unbundle != nullptr);
    assert (bundle != nullptr);
    idx_unbundle = 0;
    idx_bundle = 0;
  }

  ~pc2_batcher_t() {
    flush();
  }

  void flush() {
    std::unique_lock<std::mutex> lock(mtx);
    // Issue any partially bundles writes
    assert (idx_bundle == idx_unbundle);
    if (idx_bundle > 0) {
      while (idx_bundle < buf_to_disk_batch_t::BATCH_SIZE) {
        unbundle->batch[idx_unbundle]->size = 0;
        bundle->batch[idx_bundle] = unbundle->batch[idx_unbundle++];
        idx_bundle++;
        idx_unbundle++;
      }
      to_disk.enqueue(bundle);
      pool_empty.enqueue(unbundle);
    } else {
      // Untouched bundle/unbundle batches
      if (bundle != nullptr) {
        pool_empty.enqueue(bundle);
      }
      if (unbundle != nullptr) {
        pool_full.enqueue(bundle);
      }
    }
    bundle = nullptr;
    unbundle = nullptr;
    idx_unbundle = 0;
    idx_bundle = 0;
  }

  buf_to_disk_t<C>* dequeue() {
    std::unique_lock<std::mutex> lock(mtx);
    if (unbundle == nullptr) {
      unbundle = pool_full.dequeue();
      if (unbundle == nullptr) {
        return nullptr;
      }
    }
    buf_to_disk_t<C>* buf = unbundle->batch[idx_unbundle++];
    if (idx_unbundle == buf_to_disk_batch_t::BATCH_SIZE) {
      pool_empty.enqueue(unbundle);
      unbundle = nullptr;
      idx_unbundle = 0;
    }
    return buf;
  }

  bool enqueue(buf_to_disk_t<C>* buf) {
    std::unique_lock<std::mutex> lock(mtx);
    if (bundle == nullptr) {
      bundle = pool_empty.dequeue();
      if (bundle == nullptr) {
        //return false;
        assert(false);
      }
    }
    bundle->batch[idx_bundle++] = buf;
    if (idx_bundle == buf_to_disk_batch_t::BATCH_SIZE) {
      to_disk.enqueue(bundle);
      bundle = nullptr;
      idx_bundle = 0;
    }
    return true;
  }

  size_t size() {
    std::unique_lock<std::mutex> lock(mtx);
    return std::min
      (// Available buffer slots to store data
       (unbundle == nullptr ? 0 : (buf_to_disk_batch_t::BATCH_SIZE - idx_unbundle)) +
       pool_full.size() * buf_to_disk_batch_t::BATCH_SIZE,

                    // Available empty buffer slots
       (bundle == nullptr ? 0 : (buf_to_disk_batch_t::BATCH_SIZE - idx_bundle)) +
       pool_empty.size() * buf_to_disk_batch_t::BATCH_SIZE);
  }
};


template<class C>
void pc2_t<C>::hash_gpu(size_t partition) {
  assert (stream_count % ngpus() == 0);

  nodes_per_stream = nodes_to_read / stream_count;

  for (size_t i = 0; i < resources.size(); i++) {
    resources[i]->reset();
  }

  // Start a thread to process writes to disk
  std::atomic<bool> terminate = false;
  const size_t num_writers = (size_t)this->topology.pc2_writer_cores;
  thread_pool_t pool(num_writers);
  std::atomic<int> disk_writer_done(num_writers);
  for (size_t i = 0; i < num_writers; i++) {
    pool.spawn([this, &terminate, &disk_writer_done, i]() {
      process_writes(this->topology.pc2_writer + i, batch_size,
                     host_buf_to_disk, host_buf_pool_full,
                     terminate, disk_writer_done);
    });
  }
  pc2_batcher_t<C> disk_batcher(host_buf_pool_full, host_buf_pool_empty, host_buf_to_disk);

  bool all_done = false;
  cuda_lambda_t cuda_notify(1);
  in_ptrs_d<C::GetNumTreeRCArity()> in_d;
  buf_to_disk_t<C>* to_disk = nullptr;
  buf_to_disk_t<C>* to_disk_r = nullptr;
  fr_t* fr = nullptr;
  size_t disk_bufs_needed = 0;

  // printf("to_disk_fifo %ld, pool_full %ld, pool_empty %ld\n",
  //        host_buf_to_disk.size(), host_buf_pool_full.size(), host_buf_pool_empty.size());

  //size_t num_writes = 0;

  // auto start = std::chrono::high_resolution_clock::now();
  while (!all_done) {
    // auto now = std::chrono::high_resolution_clock::now();
    // uint64_t secs = std::chrono::duration_cast<
    //   std::chrono::seconds>(now - start).count();
    // if (secs > 60) {
    //   printf("to_disk_fifo %ld, pool_full %ld, pool_empty %ld\n",
    //          host_buf_to_disk.size(), host_buf_pool_full.size(), host_buf_pool_empty.size());
    //   for (size_t resource_num = 0; resource_num < resources.size(); resource_num++) {
    //     printf("resource %ld state %d\n", resource_num, (int)resources[resource_num]->state);
    //   }
    //   start = now;
    // }

    all_done = true;
    for (size_t resource_num = 0; resource_num < resources.size(); resource_num++) {
      gpu_resource_t<C>& resource = *resources[resource_num];
      select_gpu(resource.gpu);
      int gpu_id = resource.gpu.id();

      if (resource.state != ResourceState::DONE) {
        all_done = false;
      }

      fr_t* out_c_d = nullptr;
      fr_t* out_r_d = nullptr;
      size_t layer_offset;
      node_id_t<C> addr;
      size_t offset_c;
      size_t offset_r;
      bool write_tree_r;
      bool write_tree_c;

      // Device storage for the hash result
      if (resource.work_c.buf != nullptr) {
        out_c_d = &(*resource.work_c.buf)[0];
        out_r_d = &(*resource.work_r.buf)[0];
      }

      switch (resource.state) {
      case ResourceState::DONE:
        // Nothing
        break;

      case ResourceState::IDLE:
        // Initiate data read
        resource.last = !resource.scheduler_c.next([](work_item_t<gpu_buffer_t, C>& w) {},
                                                   &resource.work_c);
        resource.scheduler_r.next([](work_item_t<gpu_buffer_t, C>& w) {},
                                  &resource.work_r);
        if (resource.work_c.is_leaf) {
#ifdef DISABLE_FILE_READS
          resource.state = ResourceState::HASH_COLUMN;
          resource.column_data = (fr_t*)reader.get_slot(resource.id);
#else
          resource.state = ResourceState::DATA_READ;
#endif
        } else {
          resource.state = ResourceState::HASH_LEAF;
        }
        break;

      case ResourceState::DATA_READ:
        // Initiate the next data read
        resource.start_node = (// Perform batch_size nodes in parallel
                               (uint64_t)resource.work_c.idx.node() * batch_size +
                               // Each resource (GPU stream) works on a differet nodes_per_stream chunk
                               nodes_per_stream * resource.id +
                               // Each partition is size nodes_to_read
                               partition * nodes_to_read);
        resource.column_data = (fr_t*)reader.load_layers
          (resource.id,
           tree_r_only ? C::GetNumLayers() - 1 : 0, // start layer
           resource.start_node, batch_size,
           tree_r_only ? 1 : C::GetNumLayers(), // num_layers
           &resource.valid, &resource.valid_count);
        resource.state = ResourceState::DATA_WAIT;
        break;

      case ResourceState::DATA_WAIT:
        if (resource.valid.load() == resource.valid_count) {
          if (disk_batcher.size() < 1) {
            break;
          }
          to_disk = disk_batcher.dequeue();
          assert (to_disk != nullptr);

          fr_t* encode_buf = &resource.replica_data[0];

          // Copy layer 11 data to to_disk buffer for encoding/writing
          // If only building tree-r then only the last layer is present
          fr_t* layer11;
          if (tree_r_only) {
            layer11 = &resource.column_data[0];
          } else {
            layer11 = &resource.column_data[C::PARALLEL_SECTORS *
                                            (C::GetNumLayers() - 1) * batch_size];
          }
          memcpy(encode_buf, layer11,
                 C::PARALLEL_SECTORS * batch_size * sizeof(fr_t));

          // Encode non CC sectors
          for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
            if (data_files[i].is_open()) {
              for (size_t j = 0; j < batch_size; j++) {
                // Perform the field add without moving to Montgomery space
                fr_t data = data_files[i][resource.start_node + j];
                fr_t* elmt = &encode_buf[i + j * C::PARALLEL_SECTORS];
                node_t* n = (node_t*)elmt;
                if (!reader.data_is_big_endian()) {
                  n->reverse_l();
                }
                *elmt += data;
                if (!reader.data_is_big_endian()) {
                  n->reverse_l();
                }
              }
            }
          }

          // Prepare write pointers
          to_disk->size = batch_size;
          to_disk->stride = C::PARALLEL_SECTORS;
          to_disk->reverse = true;
          to_disk->offset = resource.start_node;
          for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
            to_disk->src[i] = &to_disk->data[i];
            to_disk->dst[i] = &sealed_files[i];
          }

          // Copy the encoded replica data into the disk buffer
          memcpy(&to_disk->data[0],
                 &resource.replica_data[0],
                 batch_size * C::PARALLEL_SECTORS * sizeof(fr_t));

          assert(disk_batcher.enqueue(to_disk));
          if (tree_r_only) {
            resource.state = ResourceState::HASH_COLUMN_LEAVES;
          } else {
            resource.state = ResourceState::HASH_COLUMN;
          }
        }
        break;

      case ResourceState::HASH_COLUMN:
        if (disk_batcher.size() < 1) {
          break;
        }
        to_disk = disk_batcher.dequeue();
        assert (to_disk != nullptr);

        resource.stream.HtoD(&resource.column_data_d[0], resource.column_data, resource.batch_elements);

        // Hash the columns
        switch (C::GetNumLayers()) {
          case 2:
            poseidon_columns[gpu_id].arity_2->hash_batch_device
              (out_c_d, &resource.column_data_d[0], &resource.aux_d[0],
               batch_size * C::PARALLEL_SECTORS, C::PARALLEL_SECTORS,
               resource.stream, true, false, true, true,
               !reader.data_is_big_endian());
            break;
          case 11:
            poseidon_columns[gpu_id].arity_11->hash_batch_device
              (out_c_d, &resource.column_data_d[0], &resource.aux_d[0],
               batch_size * C::PARALLEL_SECTORS, C::PARALLEL_SECTORS,
               resource.stream, true, false, true, true,
               !reader.data_is_big_endian());
            break;
        default:
          assert(false);
        }

        // Initiate copy of the hashed data from GPU
        fr = to_disk->data;
        resource.stream.DtoH(fr, out_c_d, batch_size * C::PARALLEL_SECTORS);

        // Initiate transfer of tree-c data to files
        layer_offset = layer_offsets_c[resource.work_c.idx.layer() - 1];
        addr = node_id_t<C>(resource.work_c.idx.layer() - 1,
                            resource.work_c.idx.node() * batch_size + layer_offset * resource_num);
        offset_c = tree_c_address.address(addr);
        to_disk->size = batch_size;
        to_disk->stride = 1;
        to_disk->reverse = false;
        to_disk->offset = offset_c / sizeof(fr_t);
        for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
          //to_disk->dst[i] = (fr_t*)&tree_c_files[i][partition][offset_c];
          to_disk->dst[i] = tree_c_files[i][partition];
          to_disk->src[i] = &to_disk->data[i * batch_size];
          // printf("Initiate column write[%ld] from %p to %p offset %ld size %ld\n",
          //        i, to_disk->src[i], to_disk->dst[i], to_disk->offset, to_disk->size);
        }
        //num_writes++;

        resources[resource_num]->async_done = false;
        cuda_notify.schedule(resource.stream, [this, resource_num, offset_c,
                                               to_disk, &disk_batcher]() {
          assert(disk_batcher.enqueue(to_disk));
          resources[resource_num]->async_done = true;
        });

        resource.state = ResourceState::HASH_COLUMN_LEAVES;
        break;

      case ResourceState::HASH_COLUMN_LEAVES:
        if (!resources[resource_num]->async_done) {
          break;
        }
        if (!tree_r_only) {
          if (disk_batcher.size() < 1) {
            break;
          }
          to_disk = disk_batcher.dequeue();
          assert (to_disk != nullptr);

          // Hash tree-c
          poseidon_trees[gpu_id]->hash_batch_device
            (out_c_d, out_c_d, &resource.aux_d[0],
             batch_size * C::PARALLEL_SECTORS / C::GetNumTreeRCArity(), 1,
             resource.stream, false, false, true, true,
             !reader.data_is_big_endian());
        }

        // Hash tree-r using the replica data. If there are any non-CC
        // sectors then copy the encoded replica data over
        if (has_non_cc_sectors || tree_r_only) {
          resource.stream.HtoD
            (&resource.column_data_d[batch_size * C::PARALLEL_SECTORS * (C::GetNumLayers() - 1)],
             &resource.replica_data[0], C::PARALLEL_SECTORS * batch_size);
        }
        poseidon_trees[gpu_id]->hash_batch_device
          (out_r_d,
           &resource.column_data_d[batch_size * C::PARALLEL_SECTORS * (C::GetNumLayers() - 1)],
           &resource.aux_d[0],
           batch_size * C::PARALLEL_SECTORS / C::GetNumTreeRCArity(),
           C::PARALLEL_SECTORS,
           resource.stream, false, true, true, true,
           !reader.data_is_big_endian());

        if (!tree_r_only) {
          // Initiate copy of the hashed data from GPU, reusing the host side column buffer
          resource.stream.DtoH(&to_disk->data[0], out_c_d,
                               batch_size * C::PARALLEL_SECTORS / C::GetNumTreeRCArity());

          // Initiate transfer of tree-c data to files
          layer_offset = layer_offsets_c[resource.work_c.idx.layer()];
          addr = node_id_t<C>(resource.work_c.idx.layer(),
                              resource.work_c.idx.node() * batch_size / C::GetNumTreeRCArity() +
                              layer_offset * resource_num);
          offset_c = tree_c_address.address(addr);
          to_disk->size = batch_size / C::GetNumTreeRCArity();
          to_disk->stride = 1;
          to_disk->reverse = false;
          to_disk->offset = offset_c / sizeof(fr_t);
          for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
            to_disk->dst[i] = tree_c_files[i][partition];
            to_disk->src[i] = &to_disk->data[i * batch_size / C::GetNumTreeRCArity()];
            // printf("Initiate column leaf write from %p to %p offset %ld size %ld\n",
            //        to_disk->src[i], to_disk->dst[i], to_disk->offset, to_disk->size);
          }
        }

        resources[resource_num]->async_done = false;
        cuda_notify.schedule(resource.stream, [this, resource_num, to_disk, &disk_batcher]() {
          if (!tree_r_only) {
            assert (disk_batcher.enqueue(to_disk));
          }
          resources[resource_num]->async_done = true;
        });

        resource.state = ResourceState::HASH_WAIT;
        break;

      case ResourceState::HASH_LEAF:
        write_tree_c = !tree_r_only;
        write_tree_r = resource.work_r.idx.layer() > C::GetNumTreeRDiscardRows();
        disk_bufs_needed = write_tree_c + write_tree_r;
        if (disk_batcher.size() < disk_bufs_needed) {
          break;
        }
        if (resource.last && !gpu_results_in_use.try_lock()) {
          break;
        }
        if (!tree_r_only) {
          if (write_tree_c) {
            to_disk = disk_batcher.dequeue();
            assert (to_disk != nullptr);
          }

          // Hash tree-c
          for (size_t i = 0; i < C::GetNumTreeRCArity(); i++) {
            in_d.ptrs[i] = &(*resource.work_c.inputs[i])[0];
          }

          poseidon_trees[gpu_id]->hash_batch_device_ptrs
            (out_c_d, in_d, &resource.aux_d[0],
             batch_size * C::PARALLEL_SECTORS / C::GetNumTreeRCArity(),
             C::PARALLEL_SECTORS,
             resource.stream, false, false, true, true,
             !reader.data_is_big_endian());
        }

        // Hash tree-r
        for (size_t i = 0; i < C::GetNumTreeRCArity(); i++) {
          in_d.ptrs[i] = &(*resource.work_r.inputs[i])[0];
        }
        poseidon_trees[gpu_id]->hash_batch_device_ptrs
          (out_r_d, in_d, &resource.aux_d[0],
           batch_size * C::PARALLEL_SECTORS / C::GetNumTreeRCArity(),
           C::PARALLEL_SECTORS,
           resource.stream, false, false, true, true,
           !reader.data_is_big_endian());

        if (!tree_r_only) {
          // Initiate copy of the hashed data
          resource.stream.DtoH(&to_disk->data[0], out_c_d,
                               batch_size * C::PARALLEL_SECTORS / C::GetNumTreeRCArity());
          if (resource.last) {
            // Stash the final result in a known place
            size_t stride = batch_size * C::PARALLEL_SECTORS / C::GetNumTreeRCArity();
            fr_t* host_buf_c = (fr_t*)&gpu_results_c[resource.id * stride];
            CUDA_OK(hipMemcpyAsync(host_buf_c, &to_disk->data[0],
                                    batch_size * C::PARALLEL_SECTORS / C::GetNumTreeRCArity() * sizeof(fr_t),
                                    hipMemcpyHostToHost, resource.stream));
          }

          // Compute offsets in the output files - tree-c
          layer_offset = layer_offsets_c[resource.work_c.idx.layer()];
          addr = node_id_t<C>(resource.work_c.idx.layer(),
                              resource.work_c.idx.node() * batch_size / C::GetNumTreeRCArity() +
                              layer_offset * resource_num);
          offset_c = tree_c_address.address(addr);
          to_disk->size = batch_size / C::GetNumTreeRCArity();
          to_disk->stride = 1;
          to_disk->reverse = false;
          to_disk->offset = offset_c / sizeof(fr_t);
          for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
            //to_disk->dst[i] = (fr_t*)&tree_c_files[i][partition][offset_c];
            to_disk->dst[i] = tree_c_files[i][partition];
            to_disk->src[i] = &to_disk->data[i * batch_size / C::GetNumTreeRCArity()];
            // printf("Initiate tree-c write from %p to %p offset %ld size %ld\n",
            //        to_disk->src[i], to_disk->dst[i], to_disk->offset, to_disk->size);
          }
        }

        // tree-r
        if (resource.last) {
          // Stash the final result in a known place
          size_t stride = batch_size * C::PARALLEL_SECTORS / C::GetNumTreeRCArity();
          fr_t* host_buf_r = (fr_t*)&gpu_results_r[resource.id * stride];
          resource.stream.DtoH(host_buf_r, out_r_d,
                               batch_size * C::PARALLEL_SECTORS / C::GetNumTreeRCArity());
        }

        if (write_tree_r) {
          to_disk_r = disk_batcher.dequeue();
          assert (to_disk_r != nullptr);
          resource.stream.DtoH(&to_disk_r->data[0], out_r_d,
                               batch_size * C::PARALLEL_SECTORS / C::GetNumTreeRCArity());

          layer_offset = layer_offsets_r[resource.work_r.idx.layer() - C::GetNumTreeRDiscardRows() - 1];
          addr = node_id_t<C>(resource.work_r.idx.layer() - C::GetNumTreeRDiscardRows() - 1,
                              resource.work_r.idx.node() * batch_size / C::GetNumTreeRCArity() +
                              layer_offset * resource_num);
          offset_r = tree_r_address.address(addr);
          to_disk_r->size = batch_size / C::GetNumTreeRCArity();
          to_disk_r->stride = 1;
          to_disk_r->reverse = false;
          to_disk_r->offset = offset_r / sizeof(fr_t);
          for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
            //to_disk_r->dst[i] = (fr_t*)&tree_r_files[i][partition][offset_r];
            to_disk_r->dst[i] = tree_r_files[i][partition];
            to_disk_r->src[i] = &to_disk_r->data[i * batch_size / C::GetNumTreeRCArity()];
            // printf("Initiate tree-r write from %p to %p offset %ld size %ld\n",
            //        to_disk->src[i], to_disk->dst[i], to_disk->offset, to_disk->size);
          }
        }

        // Initiate transfer of data to files
        resources[resource_num]->async_done = false;
        cuda_notify.schedule(resource.stream, [this, resource_num, &disk_batcher,
                                               to_disk, to_disk_r, write_tree_r, write_tree_c]() {
          if (resources[resource_num]->last) {
            gpu_results_in_use.unlock();
          }
          if (write_tree_c) {
            assert(disk_batcher.enqueue(to_disk));
          }
          if (write_tree_r) {
            assert(disk_batcher.enqueue(to_disk_r));
          }
          resources[resource_num]->async_done = true;
        });

        resource.state = ResourceState::HASH_WAIT;
        break;

      case ResourceState::HASH_WAIT:
        if (resource.async_done.load() == true) {
          if (resource.last) {
            resource.state = ResourceState::DONE;
          } else {
            resource.state = ResourceState::IDLE;
          }
        }
        break;

      default:
        abort();
      }
    }
  }
  for (size_t resource_num = 0; resource_num < stream_count; resource_num++) {
    resources[resource_num]->stream.sync();
  }
  disk_batcher.flush();

  terminate = true;

  // Really only need this at the last partition...
  while (disk_writer_done > 0) {}

  //printf("num_writes %ld\n", num_writes);
}

template<class C>
void pc2_t<C>::hash_cpu(fr_t* roots, size_t partition, fr_t* input,
                           std::vector<file_writer_t<fr_t>*>* tree_files,
                           size_t file_offset) {
  // This count is one layer above the leaves
  const size_t nodes_to_hash = batch_size * stream_count / C::GetNumTreeRCArity() / C::GetNumTreeRCArity();
  // Number of consecutive nodes in the input stream
  const size_t group_size = batch_size / C::GetNumTreeRCArity();
  // For simplicity of indexing require batch size to be a multiple of arity
  assert (group_size % C::GetNumTreeRCArity() == 0);

  tree_address_t<C> final_tree(nodes_to_hash, C::GetNumTreeRCArity(), sizeof(fr_t), 0);

  Poseidon hasher(C::GetNumTreeRCArity());

  auto hash_func = [this, &hasher, &final_tree, input, partition, tree_files, file_offset, group_size]
    (work_item_t<host_buffer_t, C>& w) {
    node_id_t<C> addr(w.idx.layer() - 1, w.idx.node());
    size_t offset = final_tree.address(addr) + file_offset;

    if (w.is_leaf) {
      for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
        fr_t* out = &(*w.buf)[sector];
        fr_t in[C::GetNumTreeRCArity()];

        size_t first_input_node = w.idx.node() * C::GetNumTreeRCArity();
        for (size_t i = 0; i < C::GetNumTreeRCArity(); i++) {
          size_t input_group   = (first_input_node + i) / group_size;
          size_t node_in_group = (first_input_node + i) % group_size;

          in[i] = input[input_group * group_size * C::PARALLEL_SECTORS +
                        sector * group_size + node_in_group];
        }
        hasher.Hash((uint8_t*)out, (uint8_t*)in);
        tree_files[sector][partition]->write_data(offset / sizeof(fr_t), &out[0], 1);
      }
    } else {
      for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
        fr_t* out = &(*w.buf)[sector];
        fr_t in[C::GetNumTreeRCArity()];
        for (size_t i = 0; i < C::GetNumTreeRCArity(); i++) {
          in[i] = (*w.inputs[i])[sector];
        }
        hasher.Hash((uint8_t*)out, (uint8_t*)in);
        tree_files[sector][partition]->write_data(offset / sizeof(fr_t), (fr_t*)&out[0], 1);
      }
    }
  };

  buffers_t<host_buffer_t> buffers(C::PARALLEL_SECTORS);
  scheduler_t<host_buffer_t, C> scheduler(nodes_to_hash, C::GetNumTreeRCArity(), buffers);
  host_buffer_t* host_buf = scheduler.run(hash_func);
  memcpy(roots, &(*host_buf)[0], sizeof(fr_t) * C::PARALLEL_SECTORS);
  assert (scheduler.is_done());
}

template<class C>
void pc2_t<C>::write_roots(fr_t* roots_c, fr_t* roots_r) {
  if (C::GetNumTreeRCFiles() > 1) {
    Poseidon hasher = C::GetNumTreeRCFiles() == 16 ?
                      Poseidon(2) : Poseidon(C::GetNumTreeRCFiles());
    Poseidon hasher8(8);

    for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
      fr_t in[C::GetNumTreeRCFiles()];
      fr_t out_c;
      if (!tree_r_only) {
        for (size_t i = 0; i < C::GetNumTreeRCFiles(); i++) {
          in[i] = roots_c[i * C::PARALLEL_SECTORS + sector];
        }
        if (C::GetNumTreeRCFiles() == 16) {
          hasher8.Hash((uint8_t*)&in[0], (uint8_t*)&in[0]);
          hasher8.Hash((uint8_t*)&in[1], (uint8_t*)&in[8]);
        }

        hasher.Hash((uint8_t*)&out_c, (uint8_t*)in);
      }

      fr_t out_r;
      for (size_t i = 0; i < C::GetNumTreeRCFiles(); i++) {
        in[i] = roots_r[i * C::PARALLEL_SECTORS + sector];
      }
      if (C::GetNumTreeRCFiles() == 16) {
        hasher8.Hash((uint8_t*)&in[0], (uint8_t*)&in[0]);
        hasher8.Hash((uint8_t*)&in[1], (uint8_t*)&in[8]);
      }
      hasher.Hash((uint8_t*)&out_r, (uint8_t*)in);

      int p_aux = open(p_aux_filenames[sector].c_str(), O_RDWR | O_CREAT, (mode_t)0664);
      assert (p_aux != -1);
      if (tree_r_only) {
        fr_t zero;
        zero.zero();
        assert (write(p_aux, &zero, sizeof(fr_t)) == sizeof(fr_t));
      } else {
        assert (write(p_aux, &out_c, sizeof(fr_t)) == sizeof(fr_t));
      }
      assert (write(p_aux, &out_r, sizeof(fr_t)) == sizeof(fr_t));
      close(p_aux);
    }
  } else {
    for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
      fr_t out_c = roots_c[sector];
      fr_t out_r = roots_r[sector];

      int p_aux = open(p_aux_filenames[sector].c_str(), O_RDWR | O_CREAT, (mode_t)0664);
      assert (p_aux != -1);
      if (tree_r_only) {
        fr_t zero;
        zero.zero();
        assert (write(p_aux, &zero, sizeof(fr_t)) == sizeof(fr_t));
      } else {
        assert (write(p_aux, &out_c, sizeof(fr_t)) == sizeof(fr_t));
      }
      assert (write(p_aux, &out_r, sizeof(fr_t)) == sizeof(fr_t));
      close(p_aux);
    }
  }
}

template<class C>
void pc2_hash(topology_t& topology,
              bool tree_r_only,
              streaming_node_reader_t<C>& reader,
              size_t nodes_to_read, size_t batch_size,
              size_t stream_count,
              const char** data_filenames, const char* output_dir) {
  pc2_t<C> pc2(topology, tree_r_only, reader, nodes_to_read, batch_size, stream_count,
                  data_filenames, output_dir);
  pc2.hash();
}

template<class C>
void do_pc2_cleanup(const char* output_dir) {
  std::vector<std::string> directories;
  std::vector<std::string> p_aux_filenames;
  std::vector<std::vector<std::string>> tree_c_filenames;
  std::vector<std::vector<std::string>> tree_r_filenames;
  std::vector<std::string> sealed_filenames;

  pc2_t<C>::get_filenames(output_dir,
                             directories,
                             p_aux_filenames,
                             tree_c_filenames,
                             tree_r_filenames,
                             sealed_filenames);

  for (auto fname : p_aux_filenames) {
    std::filesystem::remove(fname);
  }
  for (auto fname : sealed_filenames) {
    std::filesystem::remove(fname);
  }
  for (size_t i = 0; i < tree_c_filenames.size(); i++) {
    for (auto fname : tree_c_filenames[i]) {
      std::filesystem::remove(fname);
    }
  }
  for (size_t i = 0; i < tree_r_filenames.size(); i++) {
    for (auto fname : tree_r_filenames[i]) {
      std::filesystem::remove(fname);
    }
  }
}

#ifdef RUNTIME_SECTOR_SIZE
template void pc2_hash<sealing_config_128_2KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_128_2KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_128_4KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_128_4KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_128_16KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_128_16KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_128_32KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_128_32KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_128_8MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_128_8MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_128_16MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_128_16MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_128_1GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_128_1GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_128_64GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_128_64GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_64_2KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_64_2KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_64_4KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_64_4KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_64_16KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_64_16KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_64_32KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_64_32KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_64_8MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_64_8MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_64_16MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_64_16MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_64_1GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_64_1GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_64_64GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_64_64GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_32_2KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_32_2KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_32_4KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_32_4KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_32_16KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_32_16KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_32_32KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_32_32KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_32_8MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_32_8MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_32_16MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_32_16MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_32_1GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_32_1GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_32_64GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_32_64GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_16_2KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_16_2KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_16_4KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_16_4KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_16_16KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_16_16KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_16_32KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_16_32KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_16_8MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_16_8MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_16_16MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_16_16MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_16_1GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_16_1GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_16_64GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_16_64GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_8_2KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_8_2KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_8_4KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_8_4KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_8_16KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_8_16KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_8_32KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_8_32KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_8_8MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_8_8MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_8_16MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_8_16MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_8_1GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_8_1GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_8_64GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_8_64GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_4_2KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_4_2KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_4_4KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_4_4KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_4_16KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_4_16KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_4_32KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_4_32KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_4_8MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_4_8MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_4_16MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_4_16MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_4_1GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_4_1GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_4_64GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_4_64GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_2_2KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_2_2KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_2_4KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_2_4KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_2_16KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_2_16KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_2_32KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_2_32KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_2_8MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_2_8MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_2_16MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_2_16MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_2_1GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_2_1GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_2_64GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_2_64GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_1_2KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_1_2KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_1_4KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_1_4KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_1_16KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_1_16KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_1_32KB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_1_32KB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_1_8MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_1_8MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_1_16MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_1_16MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_1_1GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_1_1GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_1_64GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_1_64GB_t>&, size_t, size_t, size_t, const char**, const char*);
#endif
template void pc2_hash<sealing_config_128_512MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_128_512MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_128_32GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_128_32GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_64_512MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_64_512MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_64_32GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_64_32GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_32_512MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_32_512MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_32_32GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_32_32GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_16_512MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_16_512MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_16_32GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_16_32GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_8_512MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_8_512MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_8_32GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_8_32GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_4_512MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_4_512MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_4_32GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_4_32GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_2_512MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_2_512MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_2_32GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_2_32GB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_1_512MB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_1_512MB_t>&, size_t, size_t, size_t, const char**, const char*);
template void pc2_hash<sealing_config_1_32GB_t>(topology_t&, bool, streaming_node_reader_t<sealing_config_1_32GB_t>&, size_t, size_t, size_t, const char**, const char*);


#ifdef RUNTIME_SECTOR_SIZE
template void do_pc2_cleanup<sealing_config_128_2KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_128_4KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_128_16KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_128_32KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_128_8MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_128_16MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_128_1GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_128_64GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_64_2KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_64_4KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_64_16KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_64_32KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_64_8MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_64_16MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_64_1GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_64_64GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_32_2KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_32_4KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_32_16KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_32_32KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_32_8MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_32_16MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_32_1GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_32_64GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_16_2KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_16_4KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_16_16KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_16_32KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_16_8MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_16_16MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_16_1GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_16_64GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_8_2KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_8_4KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_8_16KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_8_32KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_8_8MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_8_16MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_8_1GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_8_64GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_4_2KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_4_4KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_4_16KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_4_32KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_4_8MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_4_16MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_4_1GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_4_64GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_2_2KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_2_4KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_2_16KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_2_32KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_2_8MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_2_16MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_2_1GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_2_64GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_1_2KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_1_4KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_1_16KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_1_32KB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_1_8MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_1_16MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_1_1GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_1_64GB_t>(const char* output_dir);
#endif
template void do_pc2_cleanup<sealing_config_128_512MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_128_32GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_64_512MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_64_32GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_32_512MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_32_32GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_16_512MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_16_32GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_8_512MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_8_32GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_4_512MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_4_32GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_2_512MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_2_32GB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_1_512MB_t>(const char* output_dir);
template void do_pc2_cleanup<sealing_config_1_32GB_t>(const char* output_dir);

#endif
