#include "hip/hip_runtime.h"
// g++ -g -Wall -Wextra -Werror -Wno-subobject-linkage -march=native -O3 src/tools/tree_r.cpp -Isrc/poseidon -Ideps/sppark -Ideps/blst/src -L deps/blst -lblst

// Only supports constant arity 8 throughout the tree (2KB, 32G, etc);
//
// arguments
//  last_layer_filename
// optional arguments
//  data_filename - This indicates whether or not we have a CC sector

#include <cstdint>             // uint*
#include <sys/mman.h>          // mapping
#include <sys/stat.h>          // file stats
#include <cassert>             // assertions
#include <cmath>               // log2
#include <fcntl.h>             // file open
#include <unistd.h>            // file close
#include <iostream>            // printing
#include <iomanip>             // printing
#include <chrono>              // time

#include "../pc2/cuda/pc2.cu"

#ifndef __CUDA_ARCH__
#include "../pc1/tree_r.hpp"
#include "../pc1/tree_c.hpp"
#include "../util/debug_helpers.cpp"
#include "../sealing/sector_parameters.hpp"
#include "../util/sector_util.cpp"

void usage(char* argv[]) {
  std::cout << "If no staged data file, CC is assumed" << std::endl;
  std::cout << "Usage: " << argv[0] << " [OPTIONS]" << std::endl;
  std::cout << "-h          Print help message" << std::endl;
  std::cout << "-d <path>   Staged data file" << std::endl;
  std::cout << "-i <path>   Input cache directory" << std::endl;
  std::cout << "-o <path>   Output directory" << std::endl;
  std::cout << "-c <int>    Parallel number of cores" << std::endl;
  std::cout << "-b <string> Sector size e.g 32GiB" << std::endl;
  exit(0);
}

template<class P>
void gpu_single_pc2(std::string config_filename,
                    std::string cache_dir,
                    std::string data_filename,
                    std::string output_dir) {
  topology_t topology(config_filename.c_str());
  set_core_affinity(topology.pc2_hasher);

  size_t sector_size = P::GetSectorSize();

  // Construct the layer filenames
  std::vector<std::string> layer_filenames;
  const size_t MAX = 256;
  char fname[MAX];
  const char* layer_filename_template = "%s/sc-02-data-layer-%d.dat";
  for (size_t i = 0; i < P::GetNumLayers(); i++) {
    snprintf(fname, MAX, layer_filename_template, cache_dir.c_str(), i + 1);
    layer_filenames.push_back(fname);
  }

  // Total number of streams across all GPUs
  // Use less streams if sector size is <= 16MiB
  size_t stream_count = P::GetSectorSizeLg() <= 24 ? 8 : 64;

  // Batch size in nodes. Each node includes all parallel sectors
  // Reduce batch size if sector size is <= 16MiB
  size_t batch_size = P::GetSectorSizeLg() <= 24 ? 64 * 8 : 64 * 64 * 8;

  // Nodes to read per partition
  size_t nodes_to_read = P::GetNumNodes() / P::GetNumTreeRCFiles();

  streaming_node_reader_t<sealing_config_t<1, P>> node_reader(P::GetSectorSize(), layer_filenames);

  // Allocate storage for 2x the streams to support tree-c and tree-r
  node_reader.alloc_slots(stream_count * 2, P::GetNumLayers() * batch_size, true);

  bool tree_r_only = false;
  const char* data_filenames[1];
  if (!data_filename.empty()) {
    data_filenames[0] = data_filename.c_str();
  } else {
    data_filenames[0] = nullptr;
  }
  pc2_hash<sealing_config_t<1, P>>(
    topology, tree_r_only, node_reader, nodes_to_read, batch_size,
    stream_count,data_filenames, output_dir.c_str());
}

template<class P>
void cpu_single_pc2(std::string config_filename,
                    std::string cache_dir,
                    std::string data_filename,
                    std::string output_dir,
                    std::string last_layer_filename) {


  mmap_t<node_t> p_aux_file;
  p_aux_file.mmap_write(output_dir + "/p_aux", 2 * sizeof(node_t), true);
  TreeC<P> tree_c;
  p_aux_file[0] = tree_c.BuildTreeC(cache_dir, output_dir);
  TreeR<P> tree_r;
  p_aux_file[1] = tree_r.BuildTreeR(last_layer_filename, data_filename,
                                    output_dir);
}

int main(int argc, char* argv[]) {
  int  opt   = 0;
  std::string data_filename       = "";
  std::string cache_dir           = "";
  std::string out_dir             = ".";
  std::string sector_size_string  = "";
  std::string config_filename     = "demos/rust/supra_seal.cfg";

  while ((opt = getopt(argc, argv, "c:i:d:o:b:h")) != -1) {
    switch(opt) {
      case 'c':
        std::cout << "config file               " << optarg << std::endl;
        config_filename = optarg;
        break;
      case 'i':
        std::cout << "input cache_dir           " << optarg << std::endl;
        cache_dir = optarg;
        break;
      case 'd':
        std::cout << "data_filename input       " << optarg << std::endl;
        data_filename = optarg;
        break;
      case 'o':
        std::cout << "out_dir                   " << optarg << std::endl;
        out_dir = optarg;
        break;
      case 'b':
        std::cout << "sector_size               " << optarg << std::endl;
        sector_size_string = optarg;
        break;
      case 'h':
      case ':':
      case '?':
        usage(argv);
        break;
    }
  }

  if (sector_size_string == "") {
    printf("Please specify a sector size\n");
    exit(0);
  }

  if (cache_dir.empty()) {
    printf("-c <cache_dir> must be specified\n");
    usage(argv);
  }

  size_t sector_size = get_sector_size_from_string(sector_size_string);

#ifdef __NVCC__
    // Do PC2 on the GPU if sector size is > 32KiB
  SECTOR_PARAMS_TABLE(                                                            \
    if (ngpus() && params.GetSectorSizeLg() > 15) {                               \
      gpu_single_pc2<decltype(params)>(config_filename, cache_dir, data_filename, out_dir); \
                                                                                  \
      return 0;                                                                   \
    }                                                                             \
  );
#endif
  std::string last_layer_filename = cache_dir + std::string("/sc-02-data-layer-2.dat");
  SECTOR_PARAMS_TABLE(                                                          \
    cpu_single_pc2<decltype(params)>(config_filename, cache_dir, data_filename, \
                                     out_dir, last_layer_filename);             \
  );

  return 0;
}
#endif
