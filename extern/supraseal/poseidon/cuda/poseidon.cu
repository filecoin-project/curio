// Copyright Supranational LLC

#include <util/all_gpus.cpp>
#include <ff/bls12-381.hpp>

struct kernel_params_t {
  int   t;
  int   partial_rounds;
  int   half_full_rounds;
  fr_t* round_constants;
  fr_t* mds_matrix;
  fr_t* pre_sparse_matrix;
  fr_t* sparse_matrices;
};

#include "../poseidon.hpp"
#ifndef __CUDA_ARCH__
#include "../poseidon.cpp"
#endif

// CUDA doesn't seem to like templatized kernel arguments so encapsulate
// it in a struct.
template<size_t ARITY>
struct in_ptrs_d {
  fr_t* ptrs[ARITY];
};

#include "poseidon_kernels.cu"

template<size_t ARITY_DT>
struct PoseidonInternal {
  static const size_t DOMAIN_TAG = 1;
  static const size_t ARITY = ARITY_DT - DOMAIN_TAG;

  static void hash_batch_ptrs(kernel_params_t& params, fr_t& domain_tag,
                              //fr_t* out_d, fr_t* in_d[ARITY], fr_t* aux_d,
                              fr_t* out_d, in_ptrs_d<ARITY_DT - 1> in_d, fr_t* aux_d,
                              size_t num_hashes, size_t stride,
                              const hipStream_t& stream,
                              const bool first_tree_c, const bool first_tree_r, 
                              const bool to_mont, const bool from_mont, const bool bswap,
                              const bool multi_in_ptr = true) {
    // block size for kernels 1 and 3 where we launch one thread per element
    const int block_size_13       = (256 / ARITY_DT) * ARITY_DT;
    const int hashes_per_block_13 = block_size_13 / ARITY_DT;
    // Block size for kernels 2 and 4 where we launch one thread per ARITY_DT elements
    const int block_size_24       = 128;

    int thread_count_13 = num_hashes * ARITY_DT;
    int block_count_13  = (thread_count_13 + block_size_13 - 1) / block_size_13;
    int block_count_24  = (num_hashes + block_size_24 - 1) / block_size_24;

    // printf("threads_13 %d, threads 24 %d\n",
    //        block_size_13 * block_count_13,
    //        block_size_24 * block_count_24);

    assert (aux_d != in_d.ptrs[0]);
    poseidon_hash_1_0<ARITY_DT><<<block_count_13, block_size_13,
      sizeof(fr_t) * hashes_per_block_13 * ARITY_DT, stream>>>
      (in_d, aux_d,
       domain_tag,
       params,
       num_hashes, stride,
       to_mont, bswap,
       first_tree_c, first_tree_r, multi_in_ptr);

    poseidon_hash_2<ARITY_DT><<<block_count_24, block_size_24, 0, stream>>>
      (aux_d,
       params,
       ARITY_DT * (params.half_full_rounds + 1),
       params.half_full_rounds,
       num_hashes);

    poseidon_hash_3<ARITY_DT><<<block_count_13, block_size_13,
      sizeof(fr_t) * hashes_per_block_13 * ARITY_DT, stream>>>
      (aux_d,
       params,
       ARITY_DT * (params.half_full_rounds + 1) + params.partial_rounds,
       params.half_full_rounds + params.partial_rounds,
       thread_count_13);

    poseidon_hash_4<ARITY_DT><<<block_count_24, block_size_24, 0, stream>>>
      (aux_d, out_d,
       params.mds_matrix,
       num_hashes, from_mont);
  }
};

template struct PoseidonInternal<12>;
template struct PoseidonInternal<9>;
template struct PoseidonInternal<3>;

#ifndef __CUDA_ARCH__
template<size_t ARITY_DT>
class PoseidonCuda : public Poseidon {
  static const size_t DOMAIN_TAG = 1;
  static const size_t ARITY = ARITY_DT - DOMAIN_TAG;
  
  gpu_ptr_t<fr_t> constants_d;
  kernel_params_t  kernel_params;
  const gpu_t& gpu;
  
public:
  PoseidonCuda(const gpu_t& _gpu) : Poseidon(ARITY), gpu(_gpu) {
    select_gpu(gpu);
    constants_d = gpu_ptr_t<fr_t>{(fr_t*)gpu.Dmalloc(constants_size_)};
    fr_t* constants_ptr = &constants_d[0];
    gpu.HtoD(constants_ptr, constants_file_, constants_size_ / sizeof(fr_t));
    gpu.sync();

    AssignPointers(constants_ptr,
                   &kernel_params.round_constants, &kernel_params.mds_matrix,
                   &kernel_params.pre_sparse_matrix, &kernel_params.sparse_matrices);
    kernel_params.t = t_;
    kernel_params.partial_rounds = partial_rounds_;
    kernel_params.half_full_rounds = half_full_rounds_;
  }
  
  void hash_batch(fr_t* out, fr_t* in,
                  size_t count, size_t stride,
                  const bool first_tree_c, const bool first_tree_r,
                  const bool to_mont, const bool from_mont, const bool bswap) {
    select_gpu(gpu);
    stream_t& stream = gpu[0];

    size_t batch_count = ((count + stride - 1) / stride);
    size_t elements_per_arity = batch_count * stride;
    size_t elements_to_xfer = ARITY_DT * elements_per_arity;

    dev_ptr_t<fr_t> in_d(ARITY * elements_per_arity);
    dev_ptr_t<fr_t> out_d(count);
    dev_ptr_t<fr_t> aux_d(ARITY_DT * elements_per_arity);

    // printf("elements_htod %ld element[0] %08x element[128] %08x\n",
    //        ARITY * elements_per_arity, ((uint32_t*)&in[0])[0], ((uint32_t*)&in[128])[0]);
    stream.HtoD(&in_d[0], in, ARITY * elements_per_arity);
    hash_batch_device(&out_d[0], &in_d[0], &aux_d[0],
                      count, stride,
                      stream, first_tree_c, first_tree_r,
                      to_mont, from_mont, bswap);

    stream.DtoH(out, &out_d[0], count);
    stream.sync();
  }

  void hash_batch_device(fr_t* out_d, fr_t* in_d,  fr_t* aux_d,
                         size_t count, size_t stride,
                         stream_t& stream, const bool first_tree_c, const bool first_tree_r,
                         const bool to_mont, const bool from_mont, const bool bswap) {
    select_gpu(gpu);
    in_ptrs_d<ARITY_DT - 1> in_ptrs_d;
    memset(&in_ptrs_d, 0, sizeof(in_ptrs_d));
    in_ptrs_d.ptrs[0] = in_d;
    hash_batch_device_ptrs(out_d, in_ptrs_d, aux_d,
                           count, stride,
                           stream, first_tree_c, first_tree_r,
                           to_mont, from_mont, bswap, false);
  }

  // count      - number of hash results to produce
  // The following are only used when first == true:
  // stride     - number of elements between subsequent inputs to a hash
  void hash_batch_device_ptrs(fr_t* out_d, in_ptrs_d<ARITY_DT - 1> in_d, fr_t* aux_d,
                              size_t count, size_t stride,
                              stream_t& stream, const bool first_tree_c, const bool first_tree_r,
                              const bool to_mont, const bool from_mont,
                              const bool bswap, const bool multi_in_ptrs = true) {
    select_gpu(gpu);
    assert(count % stride == 0);
    PoseidonInternal<ARITY_DT>::hash_batch_ptrs(kernel_params, domain_tag_,
                                                &out_d[0], in_d, &aux_d[0],
                                                count, stride,
                                                stream, first_tree_c, first_tree_r,
                                                to_mont, from_mont, bswap,
                                                multi_in_ptrs);
  }
};

#endif
